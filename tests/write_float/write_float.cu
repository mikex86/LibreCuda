
#include <hip/hip_runtime.h>
extern "C" __global__ void write_float_ptr(float *dst, float *src) {
    *dst = *src;
}

extern "C" __global__ void write_float_value(float *dst, float value) {
    *dst = value;
}

extern "C" __global__ void write_float_sum(float *dst, short a, float b) {
    *dst = (a + b);
}